#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define arraySize 1000

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;

	if (i < arraySize)
		c[i] = a[i] + b[i];
}

int main()
{
	int a[arraySize];
	int b[arraySize];
	int c[arraySize];

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	for (int i = 0; i < arraySize; i++) {
		a[i] = i;
		b[i] = i;
	}

	hipMalloc((void**)&dev_c, arraySize * sizeof(int));
	hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	hipMalloc((void**)&dev_b, arraySize * sizeof(int));

	hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

	addKernel KERNEL_ARGS2 (1, arraySize) (dev_c, dev_a, dev_b);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < arraySize; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return 0;
}